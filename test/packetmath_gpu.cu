#include "hip/hip_runtime.h"
// This file is part of Eigen, a lightweight C++ template library
// for linear algebra.
//
// Copyright (C) 2008-2009 Gael Guennebaud <gael.guennebaud@inria.fr>
// Copyright (C) 2006-2008 Benoit Jacob <jacob.benoit.1@gmail.com>
//
// This Source Code Form is subject to the terms of the Mozilla
// Public License v. 2.0. If a copy of the MPL was not distributed
// with this file, You can obtain one at http://mozilla.org/MPL/2.0/.

#include "gpu_common.h"
#include "packetmath_test_shared.h"
#include "random_without_cast_overflow.h"
#include "third_party/eigen3/Eigen/src/Core/GenericPacketMath.h"
#include "third_party/eigen3/Eigen/src/Core/util/XprHelper.h"

template <typename T>
inline T REF_ADD(const T& a, const T& b) {
  return a + b;
}
template <typename T>
inline T REF_SUB(const T& a, const T& b) {
  return a - b;
}
template <typename T>
inline T REF_MUL(const T& a, const T& b) {
  return a * b;
}
template <typename T>
inline T REF_DIV(const T& a, const T& b) {
  return a / b;
}
template <typename T>
inline T REF_ABS_DIFF(const T& a, const T& b) {
  return a > b ? a - b : b - a;
}

// Specializations for bool.
template <>
inline bool REF_ADD(const bool& a, const bool& b) {
  return a || b;
}
template <>
inline bool REF_SUB(const bool& a, const bool& b) {
  return a ^ b;
}
template <>
inline bool REF_MUL(const bool& a, const bool& b) {
  return a && b;
}

//==============================================================================
// Utility Functions
//==============================================================================


// Simple base class with a single empty run() method that can be statically
// conditionally overridden with SFINAE.
struct test_base {
  static void run() {};
};

//==============================================================================
// Cast
//==============================================================================

// Uses pcast to cast from one array to another.
template <typename SrcPacket, typename TgtPacket,
          typename SrcScalar = typename internal::unpacket_traits<SrcPacket>::type,
          typename TgtScalar = typename internal::unpacket_traits<TgtPacket>::type,
          int SrcCoeffRatio = internal::type_casting_traits<SrcScalar, TgtScalar>::SrcCoeffRatio,
          int TgtCoeffRatio = internal::type_casting_traits<SrcScalar, TgtScalar>::TgtCoeffRatio >
struct pcast_array;

template <typename SrcPacket, typename TgtPacket,
          typename SrcScalar, typename TgtScalar, int TgtCoeffRatio>
struct pcast_array<SrcPacket, TgtPacket, SrcScalar, TgtScalar, 1, TgtCoeffRatio> {
  static void cast(const SrcScalar* src, size_t size, TgtScalar* dst) {
    static const int SrcPacketSize = internal::unpacket_traits<SrcPacket>::size;
    static const int TgtPacketSize = internal::unpacket_traits<TgtPacket>::size;
    size_t i = 0;
    for (; i + TgtPacketSize <= size; i += TgtPacketSize) {
      internal::pstoreu(dst + i, internal::pcast<SrcPacket, TgtPacket>(internal::ploadu<SrcPacket>(src + i)));
    }
    // Leftovers that cannot be loaded into a packet.
    for (; i < size; ++i) {
      dst[i] = internal::cast<SrcScalar, TgtScalar>(src[i]);
    }
  }
};

template <typename SrcPacket, typename TgtPacket,
          typename SrcScalar, typename TgtScalar>
struct pcast_array<SrcPacket, TgtPacket, 2, 1> {
  static void cast(const SrcScalar* src, size_t size, TgtScalar* dst) {
    static const int SrcPacketSize = internal::unpacket_traits<SrcPacket>::size;
    static const int TgtPacketSize = internal::unpacket_traits<TgtPacket>::size;
    size_t i = 0;
    for (; i + TgtPacketSize <= size; i += TgtPacketSize) {
      SrcPacket a = internal::ploadu<SrcPacket>(src + i);
      SrcPacket b = internal::ploadu<SrcPacket>(src + i + SrcPacketSize);
      internal::pstoreu(dst + i, internal::pcast<SrcPacket, TgtPacket>(a, b));
    }
    // Leftovers that cannot be loaded into a packet.
    for (; i < size; ++i) {
      dst[i] = internal::cast<SrcScalar, TgtScalar>(src[i]);
    }
  }
};

template <typename SrcPacket, typename TgtPacket,
          typename SrcScalar, typename TgtScalar>
struct pcast_array<SrcPacket, TgtPacket, 4, 1> {
  static void cast(const SrcScalar* src, size_t size, TgtScalar* dst) {
    static const int SrcPacketSize = internal::unpacket_traits<SrcPacket>::size;
    static const int TgtPacketSize = internal::unpacket_traits<TgtPacket>::size;
    size_t i = 0;
    for (; i + TgtPacketSize <= size; i += TgtPacketSize) {
      SrcPacket a = internal::ploadu<SrcPacket>(src + i);
      SrcPacket b = internal::ploadu<SrcPacket>(src + i + SrcPacketSize);
      SrcPacket c = internal::ploadu<SrcPacket>(src + i + 2 * SrcPacketSize);
      SrcPacket d = internal::ploadu<SrcPacket>(src + i + 3 * SrcPacketSize);
      internal::pstoreu(dst + i, internal::pcast<SrcPacket, TgtPacket>(a, b, c, d));
    }
    // Leftovers that cannot be loaded into a packet.
    for (; i < size; ++i) {
      dst[i] = internal::cast<SrcScalar, TgtScalar>(src[i]);
    }
  }
};

template <typename SrcPacket, typename TgtPacket,
          typename SrcScalar, typename TgtScalar>
struct pcast_array<SrcPacket, TgtPacket, 8, 1> {
  static void cast(const SrcScalar* src, size_t size, TgtScalar* dst) {
    static const int SrcPacketSize = internal::unpacket_traits<SrcPacket>::size;
    static const int TgtPacketSize = internal::unpacket_traits<TgtPacket>::size;
    size_t i = 0;
    for (; i + TgtPacketSize <= size; i += TgtPacketSize) {
      SrcPacket a = internal::ploadu<SrcPacket>(src + i);
      SrcPacket b = internal::ploadu<SrcPacket>(src + i + SrcPacketSize);
      SrcPacket c = internal::ploadu<SrcPacket>(src + i + 2 * SrcPacketSize);
      SrcPacket d = internal::ploadu<SrcPacket>(src + i + 3 * SrcPacketSize);
      SrcPacket e = internal::ploadu<SrcPacket>(src + i + 4 * SrcPacketSize);
      SrcPacket f = internal::ploadu<SrcPacket>(src + i + 5 * SrcPacketSize);
      SrcPacket g = internal::ploadu<SrcPacket>(src + i + 6 * SrcPacketSize);
      SrcPacket h = internal::ploadu<SrcPacket>(src + i + 7 * SrcPacketSize);
      internal::pstoreu(dst + i, internal::pcast<SrcPacket, TgtPacket>(a, b, c, d, e, f, g, h));
    }
    // Leftovers that cannot be loaded into a packet.
    for (; i < size; ++i) {
      dst[i] = internal::cast<SrcScalar, TgtScalar>(src[i]);
    }
  }
};

template<typename SrcPacket, typename TgtPacket, int BlockSize>
struct cast_kernel {
  typedef typename Eigen::internal::unpacket_traits<SrcPacket>::type SrcScalar;
  typedef typename Eigen::internal::unpacket_traits<TgtPacket>::type TgtScalar;
  EIGEN_DEVICE_FUNC
  void operator()(int idx, const SrcScalar* in, TgtScalar* out) const {
    pcast_array<SrcPacket, TgtPacket>::run(in+idx*BlockSize, BlockSize, out+idx*BlockSize);
  }
};

template<typename SrcPacket, typename TgtPacket>
struct test_cast : test_base {
  typedef typename internal::unpacket_traits<SrcPacket>::type SrcScalar;
  typedef typename internal::unpacket_traits<TgtPacket>::type TgtScalar;
  typedef typename internal::type_casting_traits<SrcScalar, TgtScalar> TypeCastingTraits;
  static const int SrcCoeffRatio = TypeCastingTraits::SrcCoeffRatio;
  static const int TgtCoeffRatio = TypeCastingTraits::TgtCoeffRatio;
  static const int SrcPacketSize = internal::unpacket_traits<SrcPacket>::size;
  static const int TgtPacketSize = internal::unpacket_traits<TgtPacket>::size;
  static const bool HasCast =
      internal::unpacket_traits<SrcPacket>::vectorizable && internal::unpacket_traits<TgtPacket>::vectorizable &&
      TypeCastingTraits::VectorizedCast && (SrcPacketSize * SrcCoeffRatio == TgtPacketSize * TgtCoeffRatio);

  static internal::enable_if<HasCast> run() {
    static const int BlockSize = SrcPacketSize * SrcCoeffRatio;
    eigen_assert(BlockSize == TgtPacketSize * TgtCoeffRatio && "Packet sizes and cast ratios are mismatched.");

    static const int NumBlocks = 10;
    static const int DataSize = NumBlocks * BlockSize;
    Eigen::VectorX<SrcScalar> input(DataSize);
    Eigen::VectorX<TgtScalar> output(DataSize);
    Eigen::VectorX<TgtScalar> ref(DataSize);

    // Construct a packet of scalars that will not overflow when casting
    for (int i = 0; i < DataSize; ++i) {
      input[i] = internal::random_without_cast_overflow<SrcScalar, TgtScalar>::value();
      ref[i] = internal::cast<SrcScalar, TgtScalar>(input[i]);
    }

    cast_kernel<SrcPacket, TgtPacket, DataSize> kernel;
    run_on_cpu(kernel, NumBlocks, input, output);
    VERIFY_IS_APPROX(output, reference);

    run_on_gpu(kernel, NumBlocks, input, output);
    VERIFY_IS_APPROX(output, reference);
  }
};

// Runs all cast tests from SrcPacket to TgtScalar.
template <typename SrcPacket, typename TgtScalar,
          typename TgtPacket = typename internal::packet_traits<TgtScalar>::type,
          bool Vectorized = internal::packet_traits<TgtScalar>::Vectorizable,
          bool HasHalf = !internal::is_same<typename internal::unpacket_traits<TgtPacket>::half, TgtPacket>::value>
struct test_cast_runner {
  // Default to nothing for non-vectorized casts.
  static void run() {}
}

// Runs cast test from SrcPacket to TgtPacket.
template <typename SrcPacket, typename TgtScalar, typename TgtPacket>
struct test_cast_runner<SrcPacket, TgtScalar, TgtPacket, true, false> {
  static void run() { test_cast<SrcPacket, TgtPacket>::run(); }
};

// Runs cast test from SrcPacket to TgtPacket, and recursively to its
// half-packet.
template <typename SrcPacket, typename TgtScalar, typename TgtPacket>
struct test_cast_runner<SrcPacket, TgtScalar, TgtPacket, true, true> {
  static void run() {
    test_cast<SrcPacket, TgtPacket>::run();
    test_cast_runner<SrcPacket, TgtScalar, typename internal::unpacket_traits<TgtPacket>::half>::run();
  }
};

// Cast tests from real types.
template <typename Scalar, typename Packet, int IsComplex = NumTraits<Scalar>::IsComplex>
struct packetmath_pcast_ops_runner {
  static void run() {
    test_cast_runner<Packet, float>::run();
    test_cast_runner<Packet, double>::run();
    test_cast_runner<Packet, int8_t>::run();
    test_cast_runner<Packet, uint8_t>::run();
    test_cast_runner<Packet, int16_t>::run();
    test_cast_runner<Packet, uint16_t>::run();
    test_cast_runner<Packet, int32_t>::run();
    test_cast_runner<Packet, uint32_t>::run();
    test_cast_runner<Packet, int64_t>::run();
    test_cast_runner<Packet, uint64_t>::run();
    test_cast_runner<Packet, bool>::run();
    test_cast_runner<Packet, std::complex<float> >::run();
    test_cast_runner<Packet, std::complex<double> >::run();
    test_cast_runner<Packet, half>::run();
    test_cast_runner<Packet, bfloat16>::run();
  }
};

// Only some types support cast from std::complex<>.
template <typename Scalar, typename Packet>
struct packetmath_pcast_ops_runner<Scalar, Packet, /*IsComplex=*/1> {
  static void run() {
    test_cast_runner<Packet, std::complex<float> >::run();
    test_cast_runner<Packet, std::complex<double> >::run();
    test_cast_runner<Packet, half>::run();
    test_cast_runner<Packet, bfloat16>::run();
  }
};

//==============================================================================
// Boolean Mask Ops
//==============================================================================

template<typename Packet>
struct pop_kernel_base {
  typedef typename Eigen::internal::unpacket_traits<Packet>::type Scalar;
  static const int PacketSize = Eigen::internal::unpacket_traits<Packet>::size;

  virtual void operator()(int idx, const Scalar* in, Scalar* out) const {};
  virtual int num_args() const { return 0; }

  pop_kernel_base(const std::string& name) : name(name) {}
  std::string name;
};

template<typename Packet, typename POp, int NumArgs>
struct pop_kernel;

template<typename Packet, typename POp>
struct pop_kernel<Packet, POp, 1> : public pop_kernel_base<Packet> {
  pop_kernel(const std::string& name, POp pop) : pop_kernel_base(name), pop(pop) {}
  void operator()(int idx, const Scalar* in, Scalar* out) const {
    internal::pstoreu(out, pop(internal::ploadu<Packet>(in)));
  }
  int num_args() const { return 1; }
  POp pop;
};

template<typename Packet, typename POp>
struct pop_kernel<Packet, POp, 2> : public pop_kernel_base<Packet> {
  pop_kernel(const std::string& name, POp pop) : pop_kernel_base(name), pop(pop) {}
  void operator()(int idx, const Scalar* in, Scalar* out) const {
    internal::pstoreu(out, pop(internal::ploadu<Packet>(in),
                               internal::ploadu<Packet>(in + PacketSize)));
  }
  int num_args() const { return 2; }
  POp pop;
};

template<typename Packet, typename POp>
struct pop_kernel<Packet, POp, 3> : public pop_kernel_base<Packet> {
  pop_kernel(const std::string& name, POp pop) : pop_kernel_base(name), pop(pop) {}
  void operator()(int idx, const Scalar* in, Scalar* out) const {
    internal::pstoreu(out, pop(internal::ploadu<Packet>(in),
                               internal::ploadu<Packet>(in + PacketSize),
                               internal::ploadu<Packet>(in + 2*PacketSize)));
  }
  int num_args() { return 3; }
  POp pop;
};

template<typename Packet, int NumArgs, typename POp>
pop_kernel<Packet, POp, NumArgs> new_pop_kernel(const std::string& name, POp op) {
  return new pop_kernel<Packet, POp, NumArgs>(name, op);
}

template<typename Packet>
struct multi_pop_kernel {
  typedef typename Eigen::internal::unpacket_traits<Packet>::type Scalar;
  static const int PacketSize = Eigen::internal::unpacket_traits<Packet>::size;

  multi_pop_kernel(
    std::vector<pop_kernel_base<Packet>*> kernels,
    std::vector<Index> input_offsets, std::vector<Index> output_offsets) :
      kernels(kernels), input_offsets(input_offsets),
      output_offsets(output_offsets) {}

  void operator()(int idx, const Scalar* in, Scalar* out) {
    kernels[i]->operator()(idx, in + input_offsets[i], out + output_offsets[i]);
  }

  std::vector<pop_kernel_base<Packet>*> kernels;
  std::vector<Index> input_offsets;
  std::vector<Index> output_offsets;
};

#define ADD_KERNEL1_IF1COND, PACKET_OP, SCALAR_OP)                     \
if (COND) {                                                            \
  kernels.push_back(new_pop_kernel<Packet,1>(#PACKET_OP, PACKET_OP));  \
  input_offsets.push_back(input_offset);                               \
  output_offsets.push_back(output_offset);                             \
  for (Index i = 0; i<PacketSize; ++i) {                               \
    ref[i + output_offset] =                                           \
        SCALAR_OP(input[i + input_offset]);                            \
  }                                                                    \
  output_offset += PacketSize;                                         \
}

#define ADD_KERNEL2_IF(COND, PACKET_OP, SCALAR_OP)                     \
if (COND) {                                                            \
  kernels.push_back(new_pop_kernel<Packet,2>(#PACKET_OP, PACKET_OP));  \
  input_offsets.push_back(input_offset);                               \
  output_offsets.push_back(output_offset);                             \
  for (Index i = 0; i<PacketSize; ++i) {                               \
    ref[i + output_offset] =                                           \
        SCALAR_OP(input[i + input_offset],                             \
                  input[i + input_offset + PacketSize]);               \
  }                                                                    \
  output_offset += PacketSize;                                         \
}

#define ADD_KERNEL3_IF(COND, PACKET_OP, SCALAR_OP)                     \
if (COND) {                                                            \
  kernels.push_back(new_pop_kernel<Packet,3>(#PACKET_OP, PACKET_OP));  \
  input_offsets.push_back(input_offset);                               \
  output_offsets.push_back(output_offset);                             \
  for (Index i = 0; i<PacketSize; ++i) {                               \
    ref[i + output_offset] =                                           \
        SCALAR_OP(input[i + input_offset],                             \
                  input[i + input_offset + PacketSize],                \
                  input[i + input_offset + 2 * PacketSize]);           \
  }                                                                    \
  output_offset += PacketSize;                                         \
}

// Final output_offset must contain size of expected output.
template<typename Packet, typename Input, typename Ref>
void run_and_verify_kernels(
  std::vector<pop_kernel_base<Packet>*> kernels,
  std::vector<Index> input_offsets,
  std::vector<Index> output_offsets,
  const Input& input, const Ref& ref) {

  typedef typename Eigen::internal::unpacket_traits<Packet>::type Scalar;
  static const int PacketSize = Eigen::internal::unpacket_traits<Packet>::size;

  Eigen::VectorX<Scalar> cpu_output(ref.size());
  Eigen::VectorX<Scalar> gpu_output(ref.size());
  multi_pop_kernel<Packet> kernel(kernels, input_offsets, output_offsets);
  run_on_cpu(kernel, kernels.size(), input, cpu_output);
  run_on_gpu(kernel, kernels.size(), input, gpu_output);
  for (int i=0; i<kernels.size(); ++i) {
    const Index offset = output_offsets[i];
    #if !defined(EIGEN_GPU_COMPILE_PHASE)
    VERIFY_IS_APPROX(cpu_output.segment(offset, PacketSize),
                     ref.segment(offset, PacketSize));
    VERIFY_IS_APPROX(gpu_output.segment(offset, PacketSize),
                     ref.segment(offset, PacketSize));
    #endif
  }
}

template <typename Scalar, typename Packet>
void packetmath_boolean_mask_ops() {
  const int PacketSize = internal::unpacket_traits<Packet>::size;

  Index input_offset = 0;
  Index output_offset = 0;

  std::vector<Index> input_offsets;
  std::vector<Index> output_offsets;
  std::vector<pop_kernel_base<Packet>*> kernels;

  const int NumInputPackets = 6;
  const int NumOutputPackets = 8;
  Eigen::VectorX<Scalar> input(NumInputPackets * PacketSize);
  Eigen::VectorX<Scalar> ref(NumOutputPackest * PacketSize);

  // Shared random input for all tests.
  input.setRandom();

  ADD_KERNEL1_IF(true, internal::ptrue,   internal::ptrue)
  ADD_KERNEL2_IF(true, internal::pand,    internal::pand)
  ADD_KERNEL2_IF(true, internal::por,     internal::por)
  ADD_KERNEL2_IF(true, internal::pxor,    internal::pxor)
  ADD_KERNEL2_IF(true, internal::pandnot, internal::pandnot)

  // Check equality.
  input_offset += 2 * PacketSize;
  for (int i = 0; i < PacketSize; ++i) {
    input[i + input_offset] = Scalar(i + 1);
    input[i + input_offset + PacketSize] =
      internal::random<bool>() ? input[i + input_offset] : Scalar(0);
  }
  ADD_KERNEL2_IF(true, internal::pcmp_eq, internal::pcmp_eq);

  // Special cases.
  // Test (-0) == (0) for signed operations
  input_offset += 2 * PacketSize;
  for (int i = 0; i < PacketSize; ++i) {
    input[i + input_offset] = Scalar(-0);
    input[i + input_offset + PacketSize] =
      internal::random<bool>() ? input[i + input_offset] : Scalar(0);
  }
  ADD_KERNEL2_IF(NumTraits<Scalar>::IsSigned, internal::pcmp_eq, internal::pcmp_eq);

  //Test NaN
  input_offset += 2 * PacketSize;
  for (int i = 0; i < PacketSize; ++i) {
    input[i + input_offset] = std::numeric_limits<Scalar>::quiet_NaN();
    input[i + input_offset + PacketSize] =
      internal::random<bool>() ? input[i + input_offset] : Scalar(0);
  }
  ADD_KERNEL2_IF(true, internal::pcmp_eq, internal::pcmp_eq);

  // Total output size.
  output_offsets.push_back(output_offset);

  // Run and verify.
  run_and_verify_kernels<Packet>(
    kernels, input_offsets, output_offsets, input, ref);

  // cleanup
  for (int i=0; i<kernels.size(); ++i) {
    delete kernels[i];
  }

}


// Packet16b representing bool does not support ptrue, pandnot or pcmp_eq, since the scalar path
// (for some compilers) compute the bitwise and with 0x1 of the results to keep the value in [0,1].
template<>
void packetmath_boolean_mask_ops<bool, internal::packet_traits<bool>::type>() {}

template <typename Scalar, typename Packet>
void packetmath_minus_zero_add() {
  const int PacketSize = internal::unpacket_traits<Packet>::size;
  const int size = 2 * PacketSize;
  EIGEN_ALIGN_MAX Scalar data1[size];
  EIGEN_ALIGN_MAX Scalar data2[size];
  EIGEN_ALIGN_MAX Scalar ref[size];

  for (int i = 0; i < PacketSize; ++i) {
    data1[i] = Scalar(-0.0);
    data1[i + PacketSize] = Scalar(-0.0);
  }
  CHECK_CWISE2_IF(internal::packet_traits<Scalar>::HasAdd, REF_ADD, internal::padd);
}


template <typename Scalar, typename Packet>
void packetmath() {
  typedef internal::packet_traits<Scalar> PacketTraits;
  const int PacketSize = internal::unpacket_traits<Packet>::size;
  typedef typename NumTraits<Scalar>::Real RealScalar;

  if (g_first_pass)
    std::cerr << "=== Testing packet of type '" << typeid(Packet).name() << "' and scalar type '"
              << typeid(Scalar).name() << "' and size '" << PacketSize << "' ===\n";

  const int max_size = PacketSize > 4 ? PacketSize : 4;
  const int size = PacketSize * max_size;
  EIGEN_ALIGN_MAX Scalar data1[size];
  EIGEN_ALIGN_MAX Scalar data2[size];
  EIGEN_ALIGN_MAX Scalar data3[size];
  EIGEN_ALIGN_MAX Scalar ref[size];
  RealScalar refvalue = RealScalar(0);
  for (int i = 0; i < size; ++i) {
    data1[i] = internal::random<Scalar>() / RealScalar(PacketSize);
    data2[i] = internal::random<Scalar>() / RealScalar(PacketSize);
    refvalue = (std::max)(refvalue, numext::abs(data1[i]));
  }

  internal::pstore(data2, internal::pload<Packet>(data1));
  VERIFY(test::areApprox(data1, data2, PacketSize) && "aligned load/store");

  for (int offset = 0; offset < PacketSize; ++offset) {
    internal::pstore(data2, internal::ploadu<Packet>(data1 + offset));
    VERIFY(test::areApprox(data1 + offset, data2, PacketSize) && "internal::ploadu");
  }

  for (int offset = 0; offset < PacketSize; ++offset) {
    internal::pstoreu(data2 + offset, internal::pload<Packet>(data1));
    VERIFY(test::areApprox(data1, data2 + offset, PacketSize) && "internal::pstoreu");
  }

  if (internal::unpacket_traits<Packet>::masked_load_available) {
    test::packet_helper<internal::unpacket_traits<Packet>::masked_load_available, Packet> h;
    unsigned long long max_umask = (0x1ull << PacketSize);

    for (int offset = 0; offset < PacketSize; ++offset) {
      for (unsigned long long umask = 0; umask < max_umask; ++umask) {
        h.store(data2, h.load(data1 + offset, umask));
        for (int k = 0; k < PacketSize; ++k) data3[k] = ((umask & (0x1ull << k)) >> k) ? data1[k + offset] : Scalar(0);
        VERIFY(test::areApprox(data3, data2, PacketSize) && "internal::ploadu masked");
      }
    }
  }

  if (internal::unpacket_traits<Packet>::masked_store_available) {
    test::packet_helper<internal::unpacket_traits<Packet>::masked_store_available, Packet> h;
    unsigned long long max_umask = (0x1ull << PacketSize);

    for (int offset = 0; offset < PacketSize; ++offset) {
      for (unsigned long long umask = 0; umask < max_umask; ++umask) {
        internal::pstore(data2, internal::pset1<Packet>(Scalar(0)));
        h.store(data2, h.loadu(data1 + offset), umask);
        for (int k = 0; k < PacketSize; ++k) data3[k] = ((umask & (0x1ull << k)) >> k) ? data1[k + offset] : Scalar(0);
        VERIFY(test::areApprox(data3, data2, PacketSize) && "internal::pstoreu masked");
      }
    }
  }

  VERIFY((!PacketTraits::Vectorizable) || PacketTraits::HasAdd);
  VERIFY((!PacketTraits::Vectorizable) || PacketTraits::HasSub);
  VERIFY((!PacketTraits::Vectorizable) || PacketTraits::HasMul);

  CHECK_CWISE2_IF(PacketTraits::HasAdd, REF_ADD, internal::padd);
  CHECK_CWISE2_IF(PacketTraits::HasSub, REF_SUB, internal::psub);
  CHECK_CWISE2_IF(PacketTraits::HasMul, REF_MUL, internal::pmul);
  CHECK_CWISE2_IF(PacketTraits::HasDiv, REF_DIV, internal::pdiv);

  if (PacketTraits::HasNegate) CHECK_CWISE1(internal::negate, internal::pnegate);
  CHECK_CWISE1(numext::conj, internal::pconj);

  for (int offset = 0; offset < 3; ++offset) {
    for (int i = 0; i < PacketSize; ++i) ref[i] = data1[offset];
    internal::pstore(data2, internal::pset1<Packet>(data1[offset]));
    VERIFY(test::areApprox(ref, data2, PacketSize) && "internal::pset1");
  }

  {
    for (int i = 0; i < PacketSize * 4; ++i) ref[i] = data1[i / PacketSize];
    Packet A0, A1, A2, A3;
    internal::pbroadcast4<Packet>(data1, A0, A1, A2, A3);
    internal::pstore(data2 + 0 * PacketSize, A0);
    internal::pstore(data2 + 1 * PacketSize, A1);
    internal::pstore(data2 + 2 * PacketSize, A2);
    internal::pstore(data2 + 3 * PacketSize, A3);
    VERIFY(test::areApprox(ref, data2, 4 * PacketSize) && "internal::pbroadcast4");
  }

  {
    for (int i = 0; i < PacketSize * 2; ++i) ref[i] = data1[i / PacketSize];
    Packet A0, A1;
    internal::pbroadcast2<Packet>(data1, A0, A1);
    internal::pstore(data2 + 0 * PacketSize, A0);
    internal::pstore(data2 + 1 * PacketSize, A1);
    VERIFY(test::areApprox(ref, data2, 2 * PacketSize) && "internal::pbroadcast2");
  }

  VERIFY(internal::isApprox(data1[0], internal::pfirst(internal::pload<Packet>(data1))) && "internal::pfirst");

  if (PacketSize > 1) {
    // apply different offsets to check that ploaddup is robust to unaligned inputs
    for (int offset = 0; offset < 4; ++offset) {
      for (int i = 0; i < PacketSize / 2; ++i) ref[2 * i + 0] = ref[2 * i + 1] = data1[offset + i];
      internal::pstore(data2, internal::ploaddup<Packet>(data1 + offset));
      VERIFY(test::areApprox(ref, data2, PacketSize) && "ploaddup");
    }
  }

  if (PacketSize > 2) {
    // apply different offsets to check that ploadquad is robust to unaligned inputs
    for (int offset = 0; offset < 4; ++offset) {
      for (int i = 0; i < PacketSize / 4; ++i)
        ref[4 * i + 0] = ref[4 * i + 1] = ref[4 * i + 2] = ref[4 * i + 3] = data1[offset + i];
      internal::pstore(data2, internal::ploadquad<Packet>(data1 + offset));
      VERIFY(test::areApprox(ref, data2, PacketSize) && "ploadquad");
    }
  }

  ref[0] = Scalar(0);
  for (int i = 0; i < PacketSize; ++i) ref[0] += data1[i];
  VERIFY(test::isApproxAbs(ref[0], internal::predux(internal::pload<Packet>(data1)), refvalue) && "internal::predux");

  if (PacketSize == 8 && internal::unpacket_traits<typename internal::unpacket_traits<Packet>::half>::size ==
                             4)  // so far, predux_half_downto4 is only required in such a case
  {
    int HalfPacketSize = PacketSize > 4 ? PacketSize / 2 : PacketSize;
    for (int i = 0; i < HalfPacketSize; ++i) ref[i] = Scalar(0);
    for (int i = 0; i < PacketSize; ++i) ref[i % HalfPacketSize] += data1[i];
    internal::pstore(data2, internal::predux_half_dowto4(internal::pload<Packet>(data1)));
    VERIFY(test::areApprox(ref, data2, HalfPacketSize) && "internal::predux_half_dowto4");
  }

  ref[0] = Scalar(1);
  for (int i = 0; i < PacketSize; ++i) ref[0] = REF_MUL(ref[0], data1[i]);
  VERIFY(internal::isApprox(ref[0], internal::predux_mul(internal::pload<Packet>(data1))) && "internal::predux_mul");

  for (int i = 0; i < PacketSize; ++i) ref[i] = data1[PacketSize - i - 1];
  internal::pstore(data2, internal::preverse(internal::pload<Packet>(data1)));
  VERIFY(test::areApprox(ref, data2, PacketSize) && "internal::preverse");

  internal::PacketBlock<Packet> kernel;
  for (int i = 0; i < PacketSize; ++i) {
    kernel.packet[i] = internal::pload<Packet>(data1 + i * PacketSize);
  }
  ptranspose(kernel);
  for (int i = 0; i < PacketSize; ++i) {
    internal::pstore(data2, kernel.packet[i]);
    for (int j = 0; j < PacketSize; ++j) {
      VERIFY(test::isApproxAbs(data2[j], data1[i + j * PacketSize], refvalue) && "ptranspose");
    }
  }

  if (PacketTraits::HasBlend) {
    Packet thenPacket = internal::pload<Packet>(data1);
    Packet elsePacket = internal::pload<Packet>(data2);
    EIGEN_ALIGN_MAX internal::Selector<PacketSize> selector;
    for (int i = 0; i < PacketSize; ++i) {
      selector.select[i] = i;
    }

    Packet blend = internal::pblend(selector, thenPacket, elsePacket);
    EIGEN_ALIGN_MAX Scalar result[size];
    internal::pstore(result, blend);
    for (int i = 0; i < PacketSize; ++i) {
      VERIFY(test::isApproxAbs(result[i], (selector.select[i] ? data1[i] : data2[i]), refvalue));
    }
  }

  {
    for (int i = 0; i < PacketSize; ++i) {
      // "if" mask
      unsigned char v = internal::random<bool>() ? 0xff : 0;
      char* bytes = (char*)(data1 + i);
      for (int k = 0; k < int(sizeof(Scalar)); ++k) {
        bytes[k] = v;
      }
      // "then" packet
      data1[i + PacketSize] = internal::random<Scalar>();
      // "else" packet
      data1[i + 2 * PacketSize] = internal::random<Scalar>();
    }
    CHECK_CWISE3_IF(true, internal::pselect, internal::pselect);
  }

  for (int i = 0; i < size; ++i) {
    data1[i] = internal::random<Scalar>();
  }
  CHECK_CWISE1(internal::pzero, internal::pzero);
  CHECK_CWISE2_IF(true, internal::por, internal::por);
  CHECK_CWISE2_IF(true, internal::pxor, internal::pxor);
  CHECK_CWISE2_IF(true, internal::pand, internal::pand);

  packetmath_boolean_mask_ops<Scalar, Packet>();
  packetmath_pcast_ops_runner<Scalar, Packet>::run();
  packetmath_minus_zero_add<Scalar, Packet>();

  for (int i = 0; i < size; ++i) {
    data1[i] = numext::abs(internal::random<Scalar>());
  }
  CHECK_CWISE1_IF(PacketTraits::HasSqrt, numext::sqrt, internal::psqrt);
}

// Notice that this definition works for complex types as well.
// c++11 has std::log2 for real, but not for complex types.
template <typename Scalar>
Scalar log2(Scalar x) {
  return Scalar(EIGEN_LOG2E) * std::log(x);
}

template <typename Scalar, typename Packet>
void packetmath_real() {
  typedef internal::packet_traits<Scalar> PacketTraits;
  const int PacketSize = internal::unpacket_traits<Packet>::size;

  const int size = PacketSize * 4;
  EIGEN_ALIGN_MAX Scalar data1[PacketSize * 4];
  EIGEN_ALIGN_MAX Scalar data2[PacketSize * 4];
  EIGEN_ALIGN_MAX Scalar ref[PacketSize * 4];

  for (int i = 0; i < size; ++i) {
    data1[i] = Scalar(internal::random<double>(0, 1) * std::pow(10., internal::random<double>(-6, 6)));
    data2[i] = Scalar(internal::random<double>(0, 1) * std::pow(10., internal::random<double>(-6, 6)));
  }

  if (internal::random<float>(0, 1) < 0.1f) data1[internal::random<int>(0, PacketSize)] = Scalar(0);

  CHECK_CWISE1_IF(PacketTraits::HasLog, std::log, internal::plog);
  CHECK_CWISE1_IF(PacketTraits::HasLog, log2, internal::plog2);
  CHECK_CWISE1_IF(PacketTraits::HasRsqrt, 1 / std::sqrt, internal::prsqrt);

  for (int i = 0; i < size; ++i) {
    data1[i] = Scalar(internal::random<double>(-1, 1) * std::pow(10., internal::random<double>(-3, 3)));
    data2[i] = Scalar(internal::random<double>(-1, 1) * std::pow(10., internal::random<double>(-3, 3)));
  }
  CHECK_CWISE1_IF(PacketTraits::HasSin, std::sin, internal::psin);
  CHECK_CWISE1_IF(PacketTraits::HasCos, std::cos, internal::pcos);
  CHECK_CWISE1_IF(PacketTraits::HasTan, std::tan, internal::ptan);

  CHECK_CWISE1_IF(PacketTraits::HasRound, numext::round, internal::pround);
  CHECK_CWISE1_IF(PacketTraits::HasCeil, numext::ceil, internal::pceil);
  CHECK_CWISE1_IF(PacketTraits::HasFloor, numext::floor, internal::pfloor);
  CHECK_CWISE1_IF(PacketTraits::HasRint, numext::rint, internal::print);

  // See bug 1785.
  for (int i = 0; i < size; ++i) {
    data1[i] = Scalar(-1.5 + i);
    data2[i] = Scalar(-1.5 + i);
  }
  CHECK_CWISE1_IF(PacketTraits::HasRound, numext::round, internal::pround);
  CHECK_CWISE1_IF(PacketTraits::HasRint, numext::rint, internal::print);

  for (int i = 0; i < size; ++i) {
    data1[i] = Scalar(internal::random<double>(-1, 1));
    data2[i] = Scalar(internal::random<double>(-1, 1));
  }
  CHECK_CWISE1_IF(PacketTraits::HasASin, std::asin, internal::pasin);
  CHECK_CWISE1_IF(PacketTraits::HasACos, std::acos, internal::pacos);

  for (int i = 0; i < size; ++i) {
    data1[i] = Scalar(internal::random<double>(-87, 88));
    data2[i] = Scalar(internal::random<double>(-87, 88));
  }
  CHECK_CWISE1_IF(PacketTraits::HasExp, std::exp, internal::pexp);
  for (int i = 0; i < size; ++i) {
    data1[i] = Scalar(internal::random<double>(-1, 1) * std::pow(10., internal::random<double>(-6, 6)));
    data2[i] = Scalar(internal::random<double>(-1, 1) * std::pow(10., internal::random<double>(-6, 6)));
  }
  data1[0] = Scalar(1e-20);
  CHECK_CWISE1_IF(PacketTraits::HasTanh, std::tanh, internal::ptanh);
  if (PacketTraits::HasExp && PacketSize >= 2) {
    const Scalar small = std::numeric_limits<Scalar>::epsilon();
    data1[0] = std::numeric_limits<Scalar>::quiet_NaN();
    data1[1] = small;
    test::packet_helper<PacketTraits::HasExp, Packet> h;
    h.store(data2, internal::pexp(h.load(data1)));
    VERIFY((numext::isnan)(data2[0]));
    // TODO(rmlarsen): Re-enable for bfloat16.
    if (!internal::is_same<Scalar, bfloat16>::value) {
      VERIFY_IS_APPROX(std::exp(small), data2[1]);
    }

    data1[0] = -small;
    data1[1] = Scalar(0);
    h.store(data2, internal::pexp(h.load(data1)));
    // TODO(rmlarsen): Re-enable for bfloat16.
    if (!internal::is_same<Scalar, bfloat16>::value) {
      VERIFY_IS_APPROX(std::exp(-small), data2[0]);
    }
    VERIFY_IS_EQUAL(std::exp(Scalar(0)), data2[1]);

    data1[0] = (std::numeric_limits<Scalar>::min)();
    data1[1] = -(std::numeric_limits<Scalar>::min)();
    h.store(data2, internal::pexp(h.load(data1)));
    VERIFY_IS_APPROX(std::exp((std::numeric_limits<Scalar>::min)()), data2[0]);
    VERIFY_IS_APPROX(std::exp(-(std::numeric_limits<Scalar>::min)()), data2[1]);

    data1[0] = std::numeric_limits<Scalar>::denorm_min();
    data1[1] = -std::numeric_limits<Scalar>::denorm_min();
    h.store(data2, internal::pexp(h.load(data1)));
    VERIFY_IS_APPROX(std::exp(std::numeric_limits<Scalar>::denorm_min()), data2[0]);
    VERIFY_IS_APPROX(std::exp(-std::numeric_limits<Scalar>::denorm_min()), data2[1]);
  }

  if (PacketTraits::HasTanh) {
    // NOTE this test migh fail with GCC prior to 6.3, see MathFunctionsImpl.h for details.
    data1[0] = std::numeric_limits<Scalar>::quiet_NaN();
    test::packet_helper<internal::packet_traits<Scalar>::HasTanh, Packet> h;
    h.store(data2, internal::ptanh(h.load(data1)));
    VERIFY((numext::isnan)(data2[0]));
  }

  if (PacketTraits::HasExp) {
    internal::scalar_logistic_op<Scalar> logistic;
    for (int i = 0; i < size; ++i) {
      data1[i] = Scalar(internal::random<double>(-20, 20));
    }

    test::packet_helper<PacketTraits::HasExp, Packet> h;
    h.store(data2, logistic.packetOp(h.load(data1)));
    for (int i = 0; i < PacketSize; ++i) {
      VERIFY_IS_APPROX(data2[i], logistic(data1[i]));
    }
  }

#if EIGEN_HAS_C99_MATH && (__cplusplus > 199711L)
  data1[0] = std::numeric_limits<Scalar>::infinity();
  data1[1] = Scalar(-1);
  CHECK_CWISE1_IF(PacketTraits::HasLog1p, std::log1p, internal::plog1p);
  data1[0] = std::numeric_limits<Scalar>::infinity();
  data1[1] = -std::numeric_limits<Scalar>::infinity();
  CHECK_CWISE1_IF(PacketTraits::HasExpm1, std::expm1, internal::pexpm1);
#endif

  if (PacketSize >= 2) {
    data1[0] = std::numeric_limits<Scalar>::quiet_NaN();
    data1[1] = std::numeric_limits<Scalar>::epsilon();
    if (PacketTraits::HasLog) {
      test::packet_helper<PacketTraits::HasLog, Packet> h;
      h.store(data2, internal::plog(h.load(data1)));
      VERIFY((numext::isnan)(data2[0]));
      // TODO(cantonios): Re-enable for bfloat16.
      if (!internal::is_same<Scalar, bfloat16>::value) {
        VERIFY_IS_APPROX(std::log(data1[1]), data2[1]);
      }

      data1[0] = -std::numeric_limits<Scalar>::epsilon();
      data1[1] = Scalar(0);
      h.store(data2, internal::plog(h.load(data1)));
      VERIFY((numext::isnan)(data2[0]));
      VERIFY_IS_EQUAL(std::log(Scalar(0)), data2[1]);

      data1[0] = (std::numeric_limits<Scalar>::min)();
      data1[1] = -(std::numeric_limits<Scalar>::min)();
      h.store(data2, internal::plog(h.load(data1)));
      // TODO(cantonios): Re-enable for bfloat16.
      if (!internal::is_same<Scalar, bfloat16>::value) {
        VERIFY_IS_APPROX(std::log((std::numeric_limits<Scalar>::min)()), data2[0]);
      }
      VERIFY((numext::isnan)(data2[1]));

      // Note: 32-bit arm always flushes denorms to zero.
#if !EIGEN_ARCH_ARM
      if (std::numeric_limits<Scalar>::has_denorm == std::denorm_present) {
        data1[0] = std::numeric_limits<Scalar>::denorm_min();
        data1[1] = -std::numeric_limits<Scalar>::denorm_min();
        h.store(data2, internal::plog(h.load(data1)));
        // TODO(rmlarsen): Reenable.
        //        VERIFY_IS_EQUAL(std::log(std::numeric_limits<Scalar>::denorm_min()), data2[0]);
        VERIFY((numext::isnan)(data2[1]));
      }
#endif

      data1[0] = Scalar(-1.0f);
      h.store(data2, internal::plog(h.load(data1)));
      VERIFY((numext::isnan)(data2[0]));

      data1[0] = std::numeric_limits<Scalar>::infinity();
      h.store(data2, internal::plog(h.load(data1)));
      VERIFY((numext::isinf)(data2[0]));
    }
    if (PacketTraits::HasLog1p) {
      test::packet_helper<PacketTraits::HasLog1p, Packet> h;
      data1[0] = Scalar(-2);
      data1[1] = -std::numeric_limits<Scalar>::infinity();
      h.store(data2, internal::plog1p(h.load(data1)));
      VERIFY((numext::isnan)(data2[0]));
      VERIFY((numext::isnan)(data2[1]));
    }
    if (PacketTraits::HasSqrt) {
      test::packet_helper<PacketTraits::HasSqrt, Packet> h;
      data1[0] = Scalar(-1.0f);
      if (std::numeric_limits<Scalar>::has_denorm == std::denorm_present) {
        data1[1] = -std::numeric_limits<Scalar>::denorm_min();
      } else {
        data1[1] = -std::numeric_limits<Scalar>::epsilon();
      }
      h.store(data2, internal::psqrt(h.load(data1)));
      VERIFY((numext::isnan)(data2[0]));
      VERIFY((numext::isnan)(data2[1]));
    }
    // TODO(rmlarsen): Re-enable for half and bfloat16.
    if (PacketTraits::HasCos
        && !internal::is_same<Scalar, half>::value
        && !internal::is_same<Scalar, bfloat16>::value) {
      test::packet_helper<PacketTraits::HasCos, Packet> h;
      for (Scalar k = Scalar(1); k < Scalar(10000) / std::numeric_limits<Scalar>::epsilon(); k *= Scalar(2)) {
        for (int k1 = 0; k1 <= 1; ++k1) {
          data1[0] = Scalar((2 * double(k) + k1) * double(EIGEN_PI) / 2 * internal::random<double>(0.8, 1.2));
          data1[1] = Scalar((2 * double(k) + 2 + k1) * double(EIGEN_PI) / 2 * internal::random<double>(0.8, 1.2));
          h.store(data2, internal::pcos(h.load(data1)));
          h.store(data2 + PacketSize, internal::psin(h.load(data1)));
          VERIFY(data2[0] <= Scalar(1.) && data2[0] >= Scalar(-1.));
          VERIFY(data2[1] <= Scalar(1.) && data2[1] >= Scalar(-1.));
          VERIFY(data2[PacketSize + 0] <= Scalar(1.) && data2[PacketSize + 0] >= Scalar(-1.));
          VERIFY(data2[PacketSize + 1] <= Scalar(1.) && data2[PacketSize + 1] >= Scalar(-1.));

          VERIFY_IS_APPROX(data2[0], std::cos(data1[0]));
          VERIFY_IS_APPROX(data2[1], std::cos(data1[1]));
          VERIFY_IS_APPROX(data2[PacketSize + 0], std::sin(data1[0]));
          VERIFY_IS_APPROX(data2[PacketSize + 1], std::sin(data1[1]));

          VERIFY_IS_APPROX(numext::abs2(data2[0]) + numext::abs2(data2[PacketSize + 0]), Scalar(1));
          VERIFY_IS_APPROX(numext::abs2(data2[1]) + numext::abs2(data2[PacketSize + 1]), Scalar(1));
        }
      }

      data1[0] = std::numeric_limits<Scalar>::infinity();
      data1[1] = -std::numeric_limits<Scalar>::infinity();
      h.store(data2, internal::psin(h.load(data1)));
      VERIFY((numext::isnan)(data2[0]));
      VERIFY((numext::isnan)(data2[1]));

      h.store(data2, internal::pcos(h.load(data1)));
      VERIFY((numext::isnan)(data2[0]));
      VERIFY((numext::isnan)(data2[1]));

      data1[0] = std::numeric_limits<Scalar>::quiet_NaN();
      h.store(data2, internal::psin(h.load(data1)));
      VERIFY((numext::isnan)(data2[0]));
      h.store(data2, internal::pcos(h.load(data1)));
      VERIFY((numext::isnan)(data2[0]));

      data1[0] = -Scalar(0.);
      h.store(data2, internal::psin(h.load(data1)));
      VERIFY(internal::biteq(data2[0], data1[0]));
      h.store(data2, internal::pcos(h.load(data1)));
      VERIFY_IS_EQUAL(data2[0], Scalar(1));
    }
  }
}

#define CAST_CHECK_CWISE1_IF(COND, REFOP, POP, SCALAR, REFTYPE) if(COND) { \
  test::packet_helper<COND,Packet> h; \
  for (int i=0; i<PacketSize; ++i) \
    ref[i] = SCALAR(REFOP(static_cast<REFTYPE>(data1[i]))); \
  h.store(data2, POP(h.load(data1))); \
  VERIFY(test::areApprox(ref, data2, PacketSize) && #POP); \
}

template <typename Scalar>
Scalar propagate_nan_max(const Scalar& a, const Scalar& b) {
  if ((numext::isnan)(a)) return a;
  if ((numext::isnan)(b)) return b;
  return (numext::maxi)(a,b);
}

template <typename Scalar>
Scalar propagate_nan_min(const Scalar& a, const Scalar& b) {
  if ((numext::isnan)(a)) return a;
  if ((numext::isnan)(b)) return b;
  return (numext::mini)(a,b);
}

template <typename Scalar>
Scalar propagate_number_max(const Scalar& a, const Scalar& b) {
  if ((numext::isnan)(a)) return b;
  if ((numext::isnan)(b)) return a;
  return (numext::maxi)(a,b);
}

template <typename Scalar>
Scalar propagate_number_min(const Scalar& a, const Scalar& b) {
  if ((numext::isnan)(a)) return b;
  if ((numext::isnan)(b)) return a;
  return (numext::mini)(a,b);
}

template <typename Scalar, typename Packet>
void packetmath_notcomplex() {
  typedef internal::packet_traits<Scalar> PacketTraits;
  const int PacketSize = internal::unpacket_traits<Packet>::size;

  EIGEN_ALIGN_MAX Scalar data1[PacketSize * 4];
  EIGEN_ALIGN_MAX Scalar data2[PacketSize * 4];
  EIGEN_ALIGN_MAX Scalar ref[PacketSize * 4];

  Array<Scalar, Dynamic, 1>::Map(data1, PacketSize * 4).setRandom();

  VERIFY((!PacketTraits::Vectorizable) || PacketTraits::HasMin);
  VERIFY((!PacketTraits::Vectorizable) || PacketTraits::HasMax);

  CHECK_CWISE2_IF(PacketTraits::HasMin, (std::min), internal::pmin);
  CHECK_CWISE2_IF(PacketTraits::HasMax, (std::max), internal::pmax);

  CHECK_CWISE2_IF(PacketTraits::HasMin, propagate_number_min, internal::pmin<PropagateNumbers>);
  CHECK_CWISE2_IF(PacketTraits::HasMax, propagate_number_max, internal::pmax<PropagateNumbers>);
  CHECK_CWISE1(numext::abs, internal::pabs);
  CHECK_CWISE2_IF(PacketTraits::HasAbsDiff, REF_ABS_DIFF, internal::pabsdiff);

  ref[0] = data1[0];
  for (int i = 0; i < PacketSize; ++i) ref[0] = internal::pmin(ref[0], data1[i]);
  VERIFY(internal::isApprox(ref[0], internal::predux_min(internal::pload<Packet>(data1))) && "internal::predux_min");
  ref[0] = data1[0];
  for (int i = 0; i < PacketSize; ++i) ref[0] = internal::pmax(ref[0], data1[i]);
  VERIFY(internal::isApprox(ref[0], internal::predux_max(internal::pload<Packet>(data1))) && "internal::predux_max");

  for (int i = 0; i < PacketSize; ++i) ref[i] = data1[0] + Scalar(i);
  internal::pstore(data2, internal::plset<Packet>(data1[0]));
  VERIFY(test::areApprox(ref, data2, PacketSize) && "internal::plset");

  {
    unsigned char* data1_bits = reinterpret_cast<unsigned char*>(data1);
    // predux_all - not needed yet
    // for (unsigned int i=0; i<PacketSize*sizeof(Scalar); ++i) data1_bits[i] = 0xff;
    // VERIFY(internal::predux_all(internal::pload<Packet>(data1)) && "internal::predux_all(1111)");
    // for(int k=0; k<PacketSize; ++k)
    // {
    //   for (unsigned int i=0; i<sizeof(Scalar); ++i) data1_bits[k*sizeof(Scalar)+i] = 0x0;
    //   VERIFY( (!internal::predux_all(internal::pload<Packet>(data1))) && "internal::predux_all(0101)");
    //   for (unsigned int i=0; i<sizeof(Scalar); ++i) data1_bits[k*sizeof(Scalar)+i] = 0xff;
    // }

    // predux_any
    for (unsigned int i = 0; i < PacketSize * sizeof(Scalar); ++i) data1_bits[i] = 0x0;
    VERIFY((!internal::predux_any(internal::pload<Packet>(data1))) && "internal::predux_any(0000)");
    for (int k = 0; k < PacketSize; ++k) {
      for (unsigned int i = 0; i < sizeof(Scalar); ++i) data1_bits[k * sizeof(Scalar) + i] = 0xff;
      VERIFY(internal::predux_any(internal::pload<Packet>(data1)) && "internal::predux_any(0101)");
      for (unsigned int i = 0; i < sizeof(Scalar); ++i) data1_bits[k * sizeof(Scalar) + i] = 0x00;
    }
  }


  // Test NaN propagation.
  if (!NumTraits<Scalar>::IsInteger) {
    // Test reductions with no NaNs.
    ref[0] = data1[0];
    for (int i = 0; i < PacketSize; ++i) ref[0] = internal::pmin<PropagateNumbers>(ref[0], data1[i]);
    VERIFY(internal::isApprox(ref[0], internal::predux_min<PropagateNumbers>(internal::pload<Packet>(data1))) && "internal::predux_min<PropagateNumbers>");
    ref[0] = data1[0];
    for (int i = 0; i < PacketSize; ++i) ref[0] = internal::pmin<PropagateNaN>(ref[0], data1[i]);
    VERIFY(internal::isApprox(ref[0], internal::predux_min<PropagateNaN>(internal::pload<Packet>(data1))) && "internal::predux_min<PropagateNaN>");
    ref[0] = data1[0];
    for (int i = 0; i < PacketSize; ++i) ref[0] = internal::pmax<PropagateNumbers>(ref[0], data1[i]);
    VERIFY(internal::isApprox(ref[0], internal::predux_max<PropagateNumbers>(internal::pload<Packet>(data1))) && "internal::predux_max<PropagateNumbers>");
    ref[0] = data1[0];
    for (int i = 0; i < PacketSize; ++i) ref[0] = internal::pmax<PropagateNaN>(ref[0], data1[i]);
    VERIFY(internal::isApprox(ref[0], internal::predux_max<PropagateNaN>(internal::pload<Packet>(data1))) && "internal::predux_max<PropagateNumbers>");
    // A single NaN.
    const size_t index = std::numeric_limits<size_t>::quiet_NaN() % PacketSize;
    data1[index] = std::numeric_limits<Scalar>::quiet_NaN();
    VERIFY(PacketSize==1 || !(numext::isnan)(internal::predux_min<PropagateNumbers>(internal::pload<Packet>(data1))));
    VERIFY((numext::isnan)(internal::predux_min<PropagateNaN>(internal::pload<Packet>(data1))));
    VERIFY(PacketSize==1 || !(numext::isnan)(internal::predux_max<PropagateNumbers>(internal::pload<Packet>(data1))));
    VERIFY((numext::isnan)(internal::predux_max<PropagateNaN>(internal::pload<Packet>(data1))));
    // All NaNs.
    for (int i = 0; i < 4 * PacketSize; ++i) data1[i] = std::numeric_limits<Scalar>::quiet_NaN();
    VERIFY((numext::isnan)(internal::predux_min<PropagateNumbers>(internal::pload<Packet>(data1))));
    VERIFY((numext::isnan)(internal::predux_min<PropagateNaN>(internal::pload<Packet>(data1))));
    VERIFY((numext::isnan)(internal::predux_max<PropagateNumbers>(internal::pload<Packet>(data1))));
    VERIFY((numext::isnan)(internal::predux_max<PropagateNaN>(internal::pload<Packet>(data1))));

    // Test NaN propagation for coefficient-wise min and max.
    for (int i = 0; i < PacketSize; ++i) {
      data1[i] = internal::random<bool>() ? std::numeric_limits<Scalar>::quiet_NaN() : Scalar(0);
      data1[i + PacketSize] = internal::random<bool>() ? std::numeric_limits<Scalar>::quiet_NaN() : Scalar(0);
    }
    // Note: NaN propagation is implementation defined for pmin/pmax, so we do not test it here.
    CHECK_CWISE2_IF(PacketTraits::HasMin, propagate_number_min, (internal::pmin<PropagateNumbers>));
    CHECK_CWISE2_IF(PacketTraits::HasMax, propagate_number_max, internal::pmax<PropagateNumbers>);
    CHECK_CWISE2_IF(PacketTraits::HasMin, propagate_nan_min, (internal::pmin<PropagateNaN>));
    CHECK_CWISE2_IF(PacketTraits::HasMax, propagate_nan_max, internal::pmax<PropagateNaN>);
  }
}

template <typename Scalar, typename Packet, bool ConjLhs, bool ConjRhs>
void test_conj_helper(Scalar* data1, Scalar* data2, Scalar* ref, Scalar* pval) {
  const int PacketSize = internal::unpacket_traits<Packet>::size;

  internal::conj_if<ConjLhs> cj0;
  internal::conj_if<ConjRhs> cj1;
  internal::conj_helper<Scalar, Scalar, ConjLhs, ConjRhs> cj;
  internal::conj_helper<Packet, Packet, ConjLhs, ConjRhs> pcj;

  for (int i = 0; i < PacketSize; ++i) {
    ref[i] = cj0(data1[i]) * cj1(data2[i]);
    VERIFY(internal::isApprox(ref[i], cj.pmul(data1[i], data2[i])) && "conj_helper pmul");
  }
  internal::pstore(pval, pcj.pmul(internal::pload<Packet>(data1), internal::pload<Packet>(data2)));
  VERIFY(test::areApprox(ref, pval, PacketSize) && "conj_helper pmul");

  for (int i = 0; i < PacketSize; ++i) {
    Scalar tmp = ref[i];
    ref[i] += cj0(data1[i]) * cj1(data2[i]);
    VERIFY(internal::isApprox(ref[i], cj.pmadd(data1[i], data2[i], tmp)) && "conj_helper pmadd");
  }
  internal::pstore(
      pval, pcj.pmadd(internal::pload<Packet>(data1), internal::pload<Packet>(data2), internal::pload<Packet>(pval)));
  VERIFY(test::areApprox(ref, pval, PacketSize) && "conj_helper pmadd");
}

template <typename Scalar, typename Packet>
void packetmath_complex() {
  typedef internal::packet_traits<Scalar> PacketTraits;
  typedef typename Scalar::value_type RealScalar;
  const int PacketSize = internal::unpacket_traits<Packet>::size;

  const int size = PacketSize * 4;
  EIGEN_ALIGN_MAX Scalar data1[PacketSize * 4];
  EIGEN_ALIGN_MAX Scalar data2[PacketSize * 4];
  EIGEN_ALIGN_MAX Scalar ref[PacketSize * 4];
  EIGEN_ALIGN_MAX Scalar pval[PacketSize * 4];

  for (int i = 0; i < size; ++i) {
    data1[i] = internal::random<Scalar>() * Scalar(1e2);
    data2[i] = internal::random<Scalar>() * Scalar(1e2);
  }

  test_conj_helper<Scalar, Packet, false, false>(data1, data2, ref, pval);
  test_conj_helper<Scalar, Packet, false, true>(data1, data2, ref, pval);
  test_conj_helper<Scalar, Packet, true, false>(data1, data2, ref, pval);
  test_conj_helper<Scalar, Packet, true, true>(data1, data2, ref, pval);

  // Test pcplxflip.
  {
    for (int i = 0; i < PacketSize; ++i) ref[i] = Scalar(std::imag(data1[i]), std::real(data1[i]));
    internal::pstore(pval, internal::pcplxflip(internal::pload<Packet>(data1)));
    VERIFY(test::areApprox(ref, pval, PacketSize) && "pcplxflip");
  }

  if (PacketTraits::HasSqrt) {
    for (int i = 0; i < size; ++i) {
      data1[i] = Scalar(internal::random<RealScalar>(), internal::random<RealScalar>());
    }
    CHECK_CWISE1(numext::sqrt, internal::psqrt);

    // Test misc. corner cases.
    const RealScalar zero = RealScalar(0);
    const RealScalar one = RealScalar(1);
    const RealScalar inf = std::numeric_limits<RealScalar>::infinity();
    const RealScalar nan = std::numeric_limits<RealScalar>::quiet_NaN();
    data1[0] = Scalar(zero, zero);
    data1[1] = Scalar(-zero, zero);
    data1[2] = Scalar(one, zero);
    data1[3] = Scalar(zero, one);
    CHECK_CWISE1(numext::sqrt, internal::psqrt);
    data1[0] = Scalar(-one, zero);
    data1[1] = Scalar(zero, -one);
    data1[2] = Scalar(one, one);
    data1[3] = Scalar(-one, -one);
    CHECK_CWISE1(numext::sqrt, internal::psqrt);
    data1[0] = Scalar(inf, zero);
    data1[1] = Scalar(zero, inf);
    data1[2] = Scalar(-inf, zero);
    data1[3] = Scalar(zero, -inf);
    CHECK_CWISE1(numext::sqrt, internal::psqrt);
    data1[0] = Scalar(inf, inf);
    data1[1] = Scalar(-inf, inf);
    data1[2] = Scalar(inf, -inf);
    data1[3] = Scalar(-inf, -inf);
    CHECK_CWISE1(numext::sqrt, internal::psqrt);
    data1[0] = Scalar(nan, zero);
    data1[1] = Scalar(zero, nan);
    data1[2] = Scalar(nan, one);
    data1[3] = Scalar(one, nan);
    CHECK_CWISE1(numext::sqrt, internal::psqrt);
    data1[0] = Scalar(nan, nan);
    data1[1] = Scalar(inf, nan);
    data1[2] = Scalar(nan, inf);
    data1[3] = Scalar(-inf, nan);
    CHECK_CWISE1(numext::sqrt, internal::psqrt);
  }
}

template <typename Scalar, typename Packet>
void packetmath_scatter_gather() {
  typedef typename NumTraits<Scalar>::Real RealScalar;
  const int PacketSize = internal::unpacket_traits<Packet>::size;
  EIGEN_ALIGN_MAX Scalar data1[PacketSize];
  RealScalar refvalue = RealScalar(0);
  for (int i = 0; i < PacketSize; ++i) {
    data1[i] = internal::random<Scalar>() / RealScalar(PacketSize);
  }

  int stride = internal::random<int>(1, 20);

  // Buffer of zeros.
  EIGEN_ALIGN_MAX Scalar buffer[PacketSize * 20] = {};

  Packet packet = internal::pload<Packet>(data1);
  internal::pscatter<Scalar, Packet>(buffer, packet, stride);

  for (int i = 0; i < PacketSize * 20; ++i) {
    if ((i % stride) == 0 && i < stride * PacketSize) {
      VERIFY(test::isApproxAbs(buffer[i], data1[i / stride], refvalue) && "pscatter");
    } else {
      VERIFY(test::isApproxAbs(buffer[i], Scalar(0), refvalue) && "pscatter");
    }
  }

  for (int i = 0; i < PacketSize * 7; ++i) {
    buffer[i] = internal::random<Scalar>() / RealScalar(PacketSize);
  }
  packet = internal::pgather<Scalar, Packet>(buffer, 7);
  internal::pstore(data1, packet);
  for (int i = 0; i < PacketSize; ++i) {
    VERIFY(test::isApproxAbs(data1[i], buffer[i * 7], refvalue) && "pgather");
  }
}

namespace Eigen {
namespace test {

template <typename Scalar, typename PacketType>
struct runall<Scalar, PacketType, false, false> {  // i.e. float or double
  static void run() {
    packetmath<Scalar, PacketType>();
    packetmath_scatter_gather<Scalar, PacketType>();
    packetmath_notcomplex<Scalar, PacketType>();
    packetmath_real<Scalar, PacketType>();
  }
};

template <typename Scalar, typename PacketType>
struct runall<Scalar, PacketType, false, true> {  // i.e. int
  static void run() {
    packetmath<Scalar, PacketType>();
    packetmath_scatter_gather<Scalar, PacketType>();
    packetmath_notcomplex<Scalar, PacketType>();
  }
};

template <typename Scalar, typename PacketType>
struct runall<Scalar, PacketType, true, false> {  // i.e. complex
  static void run() {
    packetmath<Scalar, PacketType>();
    packetmath_scatter_gather<Scalar, PacketType>();
    packetmath_complex<Scalar, PacketType>();
  }
};

}  // namespace test
}  // namespace Eigen

EIGEN_DECLARE_TEST(packetmath_gpu) {
  ei_test_init_gpu();

  g_first_pass = true;
  for (int i = 0; i < g_repeat; i++) {
    CALL_SUBTEST_1(test::runner<float>::run());
    CALL_SUBTEST_2(test::runner<double>::run());
    CALL_SUBTEST_3(test::runner<int8_t>::run());
    CALL_SUBTEST_4(test::runner<uint8_t>::run());
    CALL_SUBTEST_5(test::runner<int16_t>::run());
    CALL_SUBTEST_6(test::runner<uint16_t>::run());
    CALL_SUBTEST_7(test::runner<int32_t>::run());
    CALL_SUBTEST_8(test::runner<uint32_t>::run());
    CALL_SUBTEST_9(test::runner<int64_t>::run());
    CALL_SUBTEST_10(test::runner<uint64_t>::run());
    CALL_SUBTEST_11(test::runner<std::complex<float> >::run());
    CALL_SUBTEST_12(test::runner<std::complex<double> >::run());
    CALL_SUBTEST_13(test::runner<half>::run());
    CALL_SUBTEST_14((packetmath<bool, internal::packet_traits<bool>::type>()));
    CALL_SUBTEST_15(test::runner<bfloat16>::run());
    g_first_pass = false;
  }
}
